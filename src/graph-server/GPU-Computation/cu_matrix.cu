#include "cu_matrix.cuh"

CuMatrix::CuMatrix( Matrix M, const hipblasHandle_t & handle_)
    :Matrix(M.getRows(),M.getCols(),M.getData())
{   
    cudaStat=hipError_t();
    handle=handle_;
    deviceMalloc();
    if(getData()!=NULL)
       deviceSetMatrix();
}

Matrix CuMatrix::getMatrix(){
    updateMatrixFromGPU();
    return Matrix(getRows(),getCols(),getData());
}

void CuMatrix::deviceMalloc(){
    unsigned rows=this->getRows();
    unsigned cols=this->getCols();
   
    cudaStat = hipMalloc ((void**)&devPtr, rows*cols*sizeof(FeatType));
    if (cudaStat != hipSuccess) {
        printf("%u\n", cudaStat);
        printf ("device memory allocation failed\n");
        exit (EXIT_FAILURE);
    }
}
void CuMatrix::deviceSetMatrix(){
    unsigned rows=this->getRows();
    unsigned cols=this->getCols();
    FeatType * data=this->getData();
    stat = hipblasSetMatrix (rows,cols, sizeof(float), data, rows , devPtr, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        switch (stat){
            case HIPBLAS_STATUS_NOT_INITIALIZED:
            printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
            break;
            case HIPBLAS_STATUS_INVALID_VALUE:
            printf("HIPBLAS_STATUS_INVALID_VALUE\n");
            break;
            case HIPBLAS_STATUS_MAPPING_ERROR:
            printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
            break;
        }
        
        hipFree (devPtr);
        hipblasDestroy(handle);
        exit (EXIT_FAILURE);
    }
}


void CuMatrix::updateMatrixFromGPU(){
    unsigned rows=this->getRows();
    unsigned cols=this->getCols();
    if(getData()==NULL)
        setData(new FeatType[getNumElemts()]);
    FeatType * data=this->getData();
    stat = hipblasGetMatrix (rows, cols, sizeof(float), devPtr, rows, data, rows );
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed\n");
        hipFree (devPtr);
        hipblasDestroy(handle);
        exit (EXIT_FAILURE);
    }
}

CuMatrix::~CuMatrix(){
    hipFree (devPtr);
}

void CuMatrix::scale(const float& alpha){
    hipblasSscal(handle, getNumElemts(),&alpha, devPtr, 1);
}

CuMatrix CuMatrix::dot(CuMatrix& B,bool A_trans,bool B_trans,float alpha,float beta){
    if(handle!=B.handle){
        std::cout<<"Handle don't match\n";
        exit(EXIT_FAILURE);
    }
    hipblasOperation_t ATrans=A_trans?HIPBLAS_OP_T:HIPBLAS_OP_N;
    hipblasOperation_t BTrans=B_trans?HIPBLAS_OP_T:HIPBLAS_OP_N;
    //1. cublas is using col-major
    //2. when cpy into/out device memory, it will do Transpose 
    //3. C=AB and C^T= (B^T*A^T)
    //This means just swap the order of multiplicaiton
    //Guide: https://peterwittek.com/cublas-matrix-c-style.html
    Matrix AT=Matrix(getCols(),getRows(),getData());
    Matrix BT=Matrix(B.getCols(),B.getRows(),B.getData());
    
    unsigned CRow=A_trans?AT.getRows():getRows();
    unsigned CCol=B_trans?BT.getCols():B.getCols();
    Matrix mat_C(CRow,CCol,(char*)NULL);//real C

    unsigned k=A_trans?getRows():getCols();
    CuMatrix C(mat_C,handle);

    stat=hipblasSgemm(handle,
        BTrans,ATrans,
        C.getCols(),C.getRows(),k,
        &alpha,
        B.devPtr,B.getCols(),
        devPtr,getCols(),
        &beta,
        C.devPtr,C.getCols());
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("SGEMM ERROR\n");
        hipFree (devPtr);
        hipblasDestroy(handle);
        exit (EXIT_FAILURE);
    }
    return C;
}

CuMatrix CuMatrix::transpose(){
    float alpha=1.0;
    float beta=0.;
    CuMatrix res(Matrix(getCols(),getRows(),(FeatType*) NULL),handle);
    hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,
        getRows(),getCols(),
        &alpha,
        devPtr,getCols(),
        &beta,
        devPtr,getCols(),
        res.devPtr,getRows());
    return res;
}
