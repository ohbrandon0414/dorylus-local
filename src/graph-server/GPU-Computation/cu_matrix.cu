#include "cu_matrix.cuh"
#include <tuple>


typedef std::tuple<int, int, EdgeType> triplet;

std::set<FeatType *> CuMatrix::MemoryPool;
CuMatrix::CuMatrix( Matrix M, const hipblasHandle_t &handle_)
    : Matrix(M.getRows(), M.getCols(), M.getData()) {
    cudaStat = hipError_t();
    handle = handle_;
    nnz = 0;
    csrVal = NULL;
    csrColInd = NULL;
    isSparse = 0;
    deviceMalloc();
    if(getData() != NULL)
        deviceSetMatrix();
}

Matrix CuMatrix::getMatrix() {
    updateMatrixFromGPU();
    return Matrix(getRows(), getCols(), getData());
}

void CuMatrix::freeGPU() {
    for(auto ptr : MemoryPool)
        hipFree (ptr);
}

void CuMatrix::loadSpCsrForward(hipsparseHandle_t &handle, unsigned numLocalVertices, std::vector<Vertex> &vertices, unsigned numGhostVertices) {
    unsigned total = numGhostVertices + numLocalVertices;
    isSparse = true;

    //GET COO FORMAT FIRST
    unsigned count = 0;
    std::set<triplet> links;
    for(auto &v : vertices) {
        links.insert(triplet(v.getLocalId(), v.getLocalId(), v.getNormFactor()));
        count++;
        for(unsigned i = 0; i <  v.getNumInEdges(); ++i) {
            InEdge &ie = v.getInEdge(i);
            if(ie.getEdgeLocation() == LOCAL_EDGE_TYPE)
                links.insert(triplet(v.getLocalId(), ie.getSourceId(), ie.getData()));
            else
                links.insert(triplet(v.getLocalId(), v.getSourceVertexLocalId(i) + numLocalVertices,  ie.getData()));
            count++;
        }
    }

    nnz = count;
    EdgeType *norms  = new EdgeType[count];
    int *rowInd = new int[count];
    int *colInd = new int[count];

    unsigned i = 0;
    for(auto link : links) {
        rowInd[i] = std::get<0>(link);
        colInd[i] = std::get<1>(link);
        norms[i] = std::get<2>(link);
        i++;
    }
    //coo device pointer
    cudaStat = hipMalloc ((void **)&csrVal, count * sizeof(EdgeType));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc ((void **)&cooRowInd, count * sizeof(int));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc ((void **)&csrColInd, count * sizeof(int));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc ((void **)&csrRowPtr, (numLocalVertices + 1) * sizeof(int));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrVal, norms, sizeof(EdgeType) * count, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(cooRowInd, rowInd, sizeof(int) * count, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrColInd, colInd, sizeof(int) * count, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);

    hipsparseXcoo2csr(handle, cooRowInd, nnz, numLocalVertices, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);


    setRows(numLocalVertices);
    setCols(total);
    delete[] rowInd;
    delete[] colInd;
    delete[] norms;
}

void CuMatrix::loadSpCsrBackward(hipsparseHandle_t &handle, unsigned numLocalVertices, std::vector<Vertex> &vertices, unsigned numGhostVertices) {
    unsigned total = numGhostVertices + numLocalVertices;
    isSparse = true;

    //GET COO FORMAT FIRST
    unsigned count = 0;
    std::set<triplet> links;
    for(auto &v : vertices) {
        links.insert(triplet(v.getLocalId(), v.getLocalId(), v.getNormFactor()));
        count++;
        for(unsigned i = 0; i <  v.getNumOutEdges(); ++i) {
            OutEdge &oe = v.getOutEdge(i);
            if(oe.getEdgeLocation() == LOCAL_EDGE_TYPE)
                links.insert(triplet(v.getLocalId(), v.getDestVertexLocalId(i), oe.getData()));
            else
                links.insert(triplet(v.getLocalId(), v.getDestVertexLocalId(i) + numLocalVertices, oe.getData()));
            count++;
        }
    }

    nnz = count;
    EdgeType *norms  = new EdgeType[count];
    int *rowInd = new int[count];
    int *colInd = new int[count];

    unsigned i = 0;
    for(auto link : links) {
        rowInd[i] = std::get<0>(link);
        colInd[i] = std::get<1>(link);
        norms[i] = std::get<2>(link);
        i++;
    }
    //coo device pointer
    cudaStat = hipMalloc ((void **)&csrVal, count * sizeof(EdgeType));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc ((void **)&cooRowInd, count * sizeof(int));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc ((void **)&csrColInd, count * sizeof(int));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc ((void **)&csrRowPtr, (numLocalVertices + 1) * sizeof(int));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrVal, norms, sizeof(EdgeType) * count, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(cooRowInd, rowInd, sizeof(int) * count, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrColInd, colInd, sizeof(int) * count, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);

    hipsparseXcoo2csr(handle, cooRowInd, nnz, numLocalVertices, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);

    setRows(numLocalVertices);
    setCols(total);
    delete[] rowInd;
    delete[] colInd;
    delete[] norms;
}

void CuMatrix::loadSpDense(FeatType *vtcsTensor, FeatType *ghostTensor,
                           unsigned numLocalVertices, unsigned numGhostVertices,
                           unsigned numFeat) {
    //Still row major
    unsigned totalVertices = (numLocalVertices + numGhostVertices);
    cudaStat = hipMalloc ((void **)&devPtr,  numFeat * sizeof(FeatType) * totalVertices);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(devPtr, vtcsTensor, sizeof(FeatType) * numLocalVertices * numFeat, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(devPtr  + numLocalVertices * numFeat, ghostTensor,
                          sizeof(FeatType) * numGhostVertices * numFeat, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    setRows(totalVertices);
    setCols(numFeat);
    MemoryPool.insert(devPtr);
}

CuMatrix CuMatrix::extractRow(unsigned row) {
    FeatType *data = getData() ? (getData() + row * getCols()) : NULL;
    CuMatrix rowVec;
    rowVec.handle = handle;
    rowVec.setData(data);
    rowVec.setRows(1);
    rowVec.setCols(getCols());
    rowVec.devPtr = devPtr + row * getCols();
    return rowVec;
}

void CuMatrix::deviceMalloc() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    cudaStat = hipMalloc ((void **)&devPtr, rows * cols * sizeof(FeatType));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed %u\n", cudaStat);
        exit (EXIT_FAILURE);
    }
    MemoryPool.insert(devPtr);
}

void CuMatrix::deviceSetMatrix() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    FeatType *data = this->getData();

    stat = hipblasSetMatrix (rows, cols, sizeof(float), data, rows, devPtr, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        switch (stat) {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
            break;
        case HIPBLAS_STATUS_INVALID_VALUE:
            printf("HIPBLAS_STATUS_INVALID_VALUE\n");
            break;
        case HIPBLAS_STATUS_MAPPING_ERROR:
            printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
            break;
        }
        hipFree (devPtr);
        hipblasDestroy(handle);
        exit (EXIT_FAILURE);
    }
}


void CuMatrix::updateMatrixFromGPU() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    if(getData() == NULL)
        setData(new FeatType[getNumElemts()]);
    FeatType *data = this->getData();
    stat = hipblasGetMatrix (rows, cols, sizeof(float), devPtr, rows, data, rows );
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed\n");
        switch (stat) {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
            break;
        case HIPBLAS_STATUS_INVALID_VALUE:
            printf("HIPBLAS_STATUS_INVALID_VALUE\n");
            break;
        case HIPBLAS_STATUS_MAPPING_ERROR:
            printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
            break;
        }
        hipFree (devPtr);
        hipblasDestroy(handle);
        exit (EXIT_FAILURE);
    }
}

CuMatrix::~CuMatrix() {}

void CuMatrix::scale(const float &alpha) {
    hipblasSscal(handle, getNumElemts(), &alpha, devPtr, 1);
}

CuMatrix CuMatrix::dot(CuMatrix &B, bool A_trans, bool B_trans, float alpha, float beta) {
    if(handle != B.handle) {
        std::cout << "Handle don't match\n";
        exit(EXIT_FAILURE);
    }
    hipblasOperation_t ATrans = A_trans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t BTrans = B_trans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    //1. cublas is using col-major
    //2. when cpy into/out device memory, it will do Transpose
    //3. C=AB and C^T= (B^T*A^T)
    //This means just swap the order of multiplicaiton
    //Guide: https://peterwittek.com/cublas-matrix-c-style.html
    Matrix AT = Matrix(getCols(), getRows(), getData());
    Matrix BT = Matrix(B.getCols(), B.getRows(), B.getData());

    unsigned CRow = A_trans ? AT.getRows() : getRows();
    unsigned CCol = B_trans ? BT.getCols() : B.getCols();
    Matrix mat_C(CRow, CCol, (char *)NULL); //real C

    unsigned k = A_trans ? getRows() : getCols();
    CuMatrix C(mat_C, handle);

    stat = hipblasSgemm(handle,
                       BTrans, ATrans,
                       C.getCols(), C.getRows(), k,
                       &alpha,
                       B.devPtr, B.getCols(),
                       devPtr, getCols(),
                       &beta,
                       C.devPtr, C.getCols());
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("SGEMM ERROR\n");
        hipFree (devPtr);
        hipblasDestroy(handle);
        exit (EXIT_FAILURE);
    }
    return C;
}

CuMatrix CuMatrix::transpose() {
    CuMatrix res(Matrix(getCols(), getRows(), (char *) NULL), handle);
    float alpha = 1.0;
    float beta = 0.;

    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                getRows(), getCols(),
                &alpha,
                devPtr, getCols(),
                &beta,
                devPtr, getCols(),
                res.devPtr, getRows());
    return res;
}
