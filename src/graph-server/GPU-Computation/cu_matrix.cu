#include "cu_matrix.cuh"
#include <tuple>


typedef std::tuple<int, int, EdgeType> triplet;

std::set<FeatType *> CuMatrix::MemoryPool;
CuMatrix::CuMatrix( Matrix M, const hipblasHandle_t &handle_)
    : Matrix(M.getRows(), M.getCols(), M.getData()) {
    cudaStat = hipError_t();
    handle = handle_;
    nnz = 0;
    csrVal = NULL;
    csrRowPtr = NULL;
    csrColInd = NULL;
    isSparse = 0;
    deviceMalloc();
    if(getData() != NULL)
        deviceSetMatrix();
}

Matrix CuMatrix::getMatrix() {
    updateMatrixFromGPU();
    return Matrix(getRows(), getCols(), getData());
}

void CuMatrix::freeGPU() {
    for(auto ptr : MemoryPool)
        hipFree (ptr);
}

void CuMatrix::loadSpCSR(hipsparseHandle_t &handle, unsigned numLocalVertices, std::vector<Vertex> &vertices, unsigned numGhostVertices) {
    unsigned total = numGhostVertices + numLocalVertices;
    isSparse = true;

    //GET COO FORMAT FIRST
    unsigned count = 0;
    std::set<triplet> links;
    for(auto &v : vertices) {
        links.insert(triplet(v.getLocalId(), v.getLocalId(), v.getNormFactor()));
        count++;
        for(unsigned i = 0; i < v.getNumInEdges(); ++i) {
            InEdge &ie = v.getInEdge(i);
            if(ie.getEdgeLocation() == LOCAL_EDGE_TYPE)
                links.insert(triplet(ie.getSourceId(), v.getLocalId(), ie.getData()));
            else
                links.insert(triplet(ie.getSourceId() + numLocalVertices, v.getLocalId(), ie.getData()));
            count++;
        }
    }
    nnz = count;
    EdgeType *norms  = new EdgeType[count];
    int *rowInd = new int[count];
    int *colInd = new int[count];

    unsigned i = 0;
    for(auto link : links) {
        rowInd[i] = std::get<0>(link);
        colInd[i] = std::get<1>(link);
        norms[i] = std::get<2>(link);
        i++;
    }

    int *cooRowInd;
    //coo device pointer
    cudaStat = hipMalloc ((void **)&csrVal, count * sizeof(EdgeType));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc ((void **)&cooRowInd, count * sizeof(int));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc ((void **)&csrColInd, count * sizeof(int));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMalloc ((void **)&csrRowPtr, (total + 1) * sizeof(int));
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrVal, norms, sizeof(EdgeType) * count, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(cooRowInd, rowInd, sizeof(int) * count, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    cudaStat = hipMemcpy(csrColInd, colInd, sizeof(int) * count, hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);
    //convert to CSR in GPU
    auto status = hipsparseXcoo2csr(handle, cooRowInd, nnz, total, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
    assert(status == HIPSPARSE_STATUS_SUCCESS);
    setRows(total);
    setCols(total);

    hipFree(cooRowInd);
    delete[] rowInd;
    delete[] colInd;
    delete[] norms;
}

void CuMatrix::loadSpDense(FeatType *vtcsTensor, FeatType *ghostTensor,
                           unsigned numLocalVertices, unsigned numGhostVertices,
                           unsigned numFeat) {
    //Still row major
    unsigned totalVertices = (numLocalVertices + numGhostVertices);
    cudaStat = hipMalloc ((void **)&devPtr,  numFeat * sizeof(FeatType) * totalVertices);
    hipMemcpy(devPtr, vtcsTensor, sizeof(FeatType)*numLocalVertices * numFeat, hipMemcpyHostToDevice);
    hipMemcpy(devPtr  + numLocalVertices * numFeat, ghostTensor,
               sizeof(FeatType)*numGhostVertices * numFeat, hipMemcpyHostToDevice);
    setRows(totalVertices);
    setCols(numFeat);
    MemoryPool.insert(devPtr);
    hipDeviceSynchronize();
}

CuMatrix CuMatrix::extractRow(unsigned row) {
    FeatType *data = getData() ? (getData() + row * getCols()) : NULL;
    CuMatrix rowVec;
    rowVec.handle = handle;
    rowVec.setData(data);
    rowVec.setRows(1);
    rowVec.setCols(getCols());
    rowVec.devPtr = devPtr + row * getCols();
    return rowVec;
}

void CuMatrix::deviceMalloc() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();

    cudaStat = hipMalloc ((void **)&devPtr, rows * cols * sizeof(FeatType));
    MemoryPool.insert(devPtr);

    if (cudaStat != hipSuccess) {
        printf("%u\n", cudaStat);
        printf ("device memory allocation failed\n");
        exit (EXIT_FAILURE);
    }
}

void CuMatrix::deviceSetMatrix() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    FeatType *data = this->getData();
    stat = hipblasSetMatrix (rows, cols, sizeof(float), data, rows, devPtr, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        switch (stat) {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
            break;
        case HIPBLAS_STATUS_INVALID_VALUE:
            printf("HIPBLAS_STATUS_INVALID_VALUE\n");
            break;
        case HIPBLAS_STATUS_MAPPING_ERROR:
            printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
            break;
        }
        hipFree (devPtr);
        hipblasDestroy(handle);
        exit (EXIT_FAILURE);
    }
}


void CuMatrix::updateMatrixFromGPU() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    if(getData() == NULL)
        setData(new FeatType[getNumElemts()]);
    FeatType *data = this->getData();
    stat = hipblasGetMatrix (rows, cols, sizeof(float), devPtr, rows, data, rows );
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed\n");
        switch (stat) {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
            break;
        case HIPBLAS_STATUS_INVALID_VALUE:
            printf("HIPBLAS_STATUS_INVALID_VALUE\n");
            break;
        case HIPBLAS_STATUS_MAPPING_ERROR:
            printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
            break;
        }
        hipFree (devPtr);
        hipblasDestroy(handle);
        exit (EXIT_FAILURE);
    }
}

CuMatrix::~CuMatrix() {}

void CuMatrix::scale(const float &alpha) {
    hipblasSscal(handle, getNumElemts(), &alpha, devPtr, 1);
}

CuMatrix CuMatrix::dot(CuMatrix &B, bool A_trans, bool B_trans, float alpha, float beta) {
    if(handle != B.handle) {
        std::cout << "Handle don't match\n";
        exit(EXIT_FAILURE);
    }
    hipblasOperation_t ATrans = A_trans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t BTrans = B_trans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    //1. cublas is using col-major
    //2. when cpy into/out device memory, it will do Transpose
    //3. C=AB and C^T= (B^T*A^T)
    //This means just swap the order of multiplicaiton
    //Guide: https://peterwittek.com/cublas-matrix-c-style.html
    Matrix AT = Matrix(getCols(), getRows(), getData());
    Matrix BT = Matrix(B.getCols(), B.getRows(), B.getData());

    unsigned CRow = A_trans ? AT.getRows() : getRows();
    unsigned CCol = B_trans ? BT.getCols() : B.getCols();
    Matrix mat_C(CRow, CCol, (char *)NULL); //real C

    unsigned k = A_trans ? getRows() : getCols();
    CuMatrix C(mat_C, handle);

    stat = hipblasSgemm(handle,
                       BTrans, ATrans,
                       C.getCols(), C.getRows(), k,
                       &alpha,
                       B.devPtr, B.getCols(),
                       devPtr, getCols(),
                       &beta,
                       C.devPtr, C.getCols());
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("SGEMM ERROR\n");
        hipFree (devPtr);
        hipblasDestroy(handle);
        exit (EXIT_FAILURE);
    }
    return C;
}

CuMatrix CuMatrix::transpose() {
    float alpha = 1.0;
    float beta = 0.;
    CuMatrix res(Matrix(getCols(), getRows(), (FeatType *) NULL), handle);
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
                getRows(), getCols(),
                &alpha,
                devPtr, getCols(),
                &beta,
                devPtr, getCols(),
                res.devPtr, getRows());
    return res;
}
